#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "utils.h"
#include "wavefront.cuh"
#include "kernels.cuh"

#define EWAVEFRONT_DIAGONAL(h,v) ((h)-(v))
#define EWAVEFRONT_OFFSET(h,v)   (h)

// TODO: Check max_gpu_size to take in accont the multiple arrays stored on GPU
bool Sequences::GPU_memory_init () {
    CLOCK_INIT()
    // Send patterns to GPU
    size_t req_memory = this->batch_size * sizeof(WF_element);
    if (req_memory > MAX_GPU_SIZE) {
        WF_ERROR("Required memory is bigger than available memory in GPU");
        return false;
    }

    // Start the clock for benchmanrk purposes if DEBUG_MODE is enabled
    CLOCK_START()

    hipMalloc((void **) &(this->d_elements), req_memory);
    CUDA_CHECK_ERR;

    // Allocate a big chunk of memory only once for the different packed
    // patterns and texts
    SEQ_TYPE* base_ptr;
    size_t seq_size_bytes = this->sequences_reader.max_seq_len * sizeof(SEQ_TYPE);
    // * 2 sequences per element (pattern and text)
    hipMalloc((void **) &(base_ptr),
               (seq_size_bytes * 2) * this->batch_size);
    CUDA_CHECK_ERR;
    hipMemset(base_ptr, 0, (seq_size_bytes * 2) * this->batch_size);
    CUDA_CHECK_ERR;
    this->sequences_device_ptr = base_ptr;
    DEBUG("Allocating memory to store packed text/patterns on device (%zu MiB)",
          (seq_size_bytes * 2 * this->batch_size) / (1 << 20));


    // Allocate a big chunk of memory only once for the different patterns and
    // texts
    SEQ_TYPE* base_ptr_unpacked;
    seq_size_bytes = this->sequences_reader.max_seq_len_unpacked * sizeof(SEQ_TYPE);
    // * 2 sequences per element (pattern and text)
    hipMalloc((void **) &(base_ptr_unpacked),
               (seq_size_bytes * 2) * this->batch_size);
    CUDA_CHECK_ERR;
    this->sequences_device_ptr_unpacked = base_ptr_unpacked;
    DEBUG("Allocating memory to store unpacked text/patterns on device (%zu MiB)",
          (seq_size_bytes * 2 * this->batch_size) / (1 << 20));

    // Copy all the unpacked secuences for this batch to the fresh allocated
    // memory on GPU
    SEQ_TYPE* initial_seq_ptr =
            this->sequences_reader.get_sequences_buffer_unpacked() +
            (this->sequences_reader.max_seq_len_unpacked * 2 * this->initial_alignment);
    hipMemcpy(base_ptr_unpacked, initial_seq_ptr,
               (seq_size_bytes * 2) * this->batch_size,
               hipMemcpyHostToDevice);
    CUDA_CHECK_ERR;

    dim3 gridSize(this->batch_size);
    // Max block size is 1024 threads, so, even if the sequence length is more
    // than 512, we are limited to use 512 threads for pattern, and another 512
    // for the text.
    dim3 blockSize(min(this->sequences_reader.seq_len/4, 256L), 2);
    DEBUG("Lauching sequences packing kernel on GPU with grid(%d) and block(%d, %d).",
          gridSize.x, blockSize.x, blockSize.y);
    // Unpacked pattern+text in shared memory
    int shmem_size = this->sequences_reader.max_seq_len_unpacked * 2;
    compact_sequences<<<gridSize, blockSize, shmem_size>>>(
                                        this->sequences_device_ptr_unpacked,
                                        this->sequences_device_ptr,
                                        this->sequences_reader.max_seq_len_unpacked,
                                        this->sequences_reader.max_seq_len);

    hipStreamSynchronize(0);
    CUDA_CHECK_ERR;
    DEBUG("Sequences packed.");

    hipMemcpy(this->d_elements, &this->elements[this->initial_alignment],
               this->batch_size * sizeof(WF_element), hipMemcpyHostToDevice);
    CUDA_CHECK_ERR;

#ifdef DEBUG_MODE
    size_t total_memory  = req_memory + seq_size_bytes * 2 * this->batch_size;
    DEBUG("GPU pattern/text memory initialized, %zu MiB used.",
          total_memory / (1 << 20));
    CLOCK_STOP("GPU pattern/text memory initializaion.")
#endif

    return true;
}

bool Sequences::GPU_memory_free () {
    DEBUG("Freeing GPU memory.");
    CLOCK_INIT()
    CLOCK_START()
    // Free all the texts/patterns
    hipFree(this->sequences_device_ptr);
    hipFree(this->sequences_device_ptr_unpacked);
    CUDA_CHECK_ERR;
    hipFree(this->d_elements);
    CUDA_CHECK_ERR;

    CLOCK_STOP("Text/patterns GPU memory freed.")

    DEBUG("GPU memory freed.")
    return true;
}

bool Sequences::GPU_prepare_memory_next_batch () {
    // first "alginment" (sequence pair) of the current batch
    int curr_position = (++this->batch_idx * this->batch_size);
    if (curr_position >= this->num_elements) {
        DEBUG("All batches have already been processed.");
        return false;
    }
    DEBUG("Rearranging memory for batch iteration %d (position %d)",
           this->batch_idx, initial_alignment + curr_position);
    // The last "batch" may be sorter than a complete batch, e.g 10 elements,
    // batch size of 3
    int curr_batch_size = ((this->num_elements - curr_position) < this->batch_size) ?
                            (this->num_elements - curr_position) : this->batch_size;

    // Send the new unpacked text/pattern sequences to device
    size_t seq_size_bytes =
        this->sequences_reader.max_seq_len_unpacked * sizeof(SEQ_TYPE);
    SEQ_TYPE* first_pos_ptr = PATTERN_PTR(
        this->elements[curr_position + this->initial_alignment].alignment_idx,
        this->sequences_reader.get_sequences_buffer_unpacked(),
        this->sequences_reader.max_seq_len_unpacked);

    hipMemcpyAsync(this->sequences_device_ptr_unpacked,
               first_pos_ptr,
               (seq_size_bytes * 2) * curr_batch_size,
               hipMemcpyHostToDevice,
               this->HtD_stream);
    CUDA_CHECK_ERR

    // Zero the packed sequences
    hipMemsetAsync(this->sequences_device_ptr,
                    0,
                    this->sequences_reader.max_seq_len * 2 * this->batch_size,
                    this->HtD_stream);


    dim3 gridSize(this->batch_size);
    // Max block size is 1024 threads, so, even if the sequence length is more
    // than 512, we are limited to use 512 threads for pattern, and another 512
    // for the text.
    dim3 blockSize(min(this->sequences_reader.seq_len/4, 256L), 2);
    // Pack the sequences on GPU
    compact_sequences<<<gridSize, blockSize, 0, this->HtD_stream>>>(
                                        this->sequences_device_ptr_unpacked,
                                        this->sequences_device_ptr,
                                        this->sequences_reader.max_seq_len_unpacked,
                                        this->sequences_reader.max_seq_len);

    // Send the new text_len and pattern_len to device
    hipMemcpyAsync(this->d_elements, &this->elements[curr_position + initial_alignment],
               curr_batch_size * sizeof(WF_element), hipMemcpyHostToDevice,
               this->HtD_stream);
    CUDA_CHECK_ERR;

    return true;
}

void Sequences::GPU_launch_wavefront_distance () {
    // TODO: Determine better the number of threads
    int threads_x = 64;

    int blocks_x;
    // Check if the current batch is smaller than "batch_size"
    size_t sequences_remaining = this->num_elements - this->batch_size *
                                 this->batch_idx;
    if (sequences_remaining < this->batch_size)
        blocks_x = sequences_remaining;
    else
        blocks_x = this->batch_size;
    blocks_x = (blocks_x > MAX_BLOCKS) ?  MAX_BLOCKS : blocks_x;

    dim3 numBlocks(blocks_x, 1);
    dim3 blockDim(threads_x, 1);

    // text + pattern with allowance of 100% error
    int shared_mem = this->sequences_reader.max_seq_len * 2
                     // 2 complete wavefronts, add 2 to the number of elements
                     // in a wavefront to avoid loop peeling
                     + 2 * (WF_ELEMENTS(this->max_distance) + 2) * sizeof(ewf_offset_t);

    // Wait until the sequences are copied to the device
    hipStreamSynchronize(this->HtD_stream);
    CUDA_CHECK_ERR

    DEBUG("Launching wavefront alignment on GPU. %d elements with %d blocks "
          "of %d threads, and %d KiB of shared memory", blocks_x, blocks_x,
          threads_x, shared_mem / (1 << 10));

    WF_edit_distance<<<numBlocks, blockDim, shared_mem>>>(this->d_elements,
                                              this->sequences_device_ptr,
                                              this->max_distance,
                                              this->sequences_reader.max_seq_len,
                                              this->d_cigars);
#ifdef DEBUG_MODE
    // CopyIn copies the packed backtraces in
    size_t curr_position = (this->batch_idx * this->batch_size) +
                        this->initial_alignment;
    this->h_cigars.copyIn(this->d_cigars);
    SEQ_TYPE* seq_base_ptr = this->sequences_reader.get_sequences_buffer_unpacked();
    size_t max_seq_len = this->sequences_reader.max_seq_len_unpacked;
    int total_corrects = 0;
    for (int i=0; i<blocks_x; i++) {
        if (this->h_cigars.check_cigar(i, this->elements[curr_position + i],
            seq_base_ptr, max_seq_len))
            total_corrects++;
    }

    if (total_corrects == blocks_x)
        DEBUG_GREEN("Correct alignments: %d/%d", total_corrects, blocks_x)
    else
        DEBUG_RED("Correct alignments: %d/%d", total_corrects, blocks_x)
#endif
}

// Returns false when everything is comple
bool Sequences::prepare_next_batch () {
    // Wait for the kernel to finish
    hipStreamSynchronize(0);
    CUDA_CHECK_ERR;
    bool ret;

    // This is async
    ret = this->GPU_prepare_memory_next_batch();

    // This is sync
    this->h_cigars.copyIn(this->d_cigars);

    // Put all the device cigars at 0 again
    //this->d_cigars.device_reset();
    this->h_cigars.reset();

    return ret;
}
