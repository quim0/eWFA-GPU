#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "utils.h"
#include "wavefront.h"

// TODO: Check max_gpu_size to take in accont the multiple arrays stored on GPU
bool Sequences::GPU_memory_init () {
    CLOCK_INIT()
    // Send patterns to GPU
    size_t req_memory = this->num_elements * sizeof(WF_element);
    if (req_memory > MAX_GPU_SIZE) {
        WF_ERROR("Required memory is bigger than available memory in GPU");
        return false;
    }

    // Start the clock for benchmanrk purposes if DEBUG_MODE is enabled
    CLOCK_START()

    hipMalloc((void **) &(this->d_elements), req_memory);
    CUDA_CHECK_ERR;

    // Allocate a big chunk of memory only once for the different patterns and
    // texts
    uint8_t* base_ptr;
    size_t seq_size_bytes = this->sequence_len * sizeof(SEQ_TYPE);
    // *2 sequences per element (pattern and text)
    hipMalloc((void **) &(base_ptr),
               (seq_size_bytes * 2) * this->num_elements);
    CUDA_CHECK_ERR;
    DEBUG("Allocating memory to store text/patterns on device (%zu bytes)",
          seq_size_bytes * 2 * this->num_elements);

    // Copy all the secuences to the fresh allocated memory on GPU
    hipMemcpy(base_ptr, this->elements[0].text,
               (seq_size_bytes * 2) * this->num_elements,
               hipMemcpyHostToDevice);
    CUDA_CHECK_ERR;

    // TODO: Create tmp array of elements in host and do just on memcpy
    // Create a temporary array o WF_elements in host memory, store all the
    // device pointers here and then do a single memcpy. This is done to avoid
    // doing a hipMemcpy on each iteration of the loop, which dramatically
    // drops the performance.
    WF_element *tmp_wf_elements_host = (WF_element*)
                                calloc(this->num_elements, sizeof(WF_element));

    // += 2 because every element have two sequences (pattern and text)
    for (int i=0; i<this->num_elements; i += 2) {
        WF_element *tmp_host_elem = &tmp_wf_elements_host[i / 2];
        SEQ_TYPE* seq1 = (SEQ_TYPE*)(base_ptr + i * seq_size_bytes);
        SEQ_TYPE* seq2 = (SEQ_TYPE*)(base_ptr + (i + 1) * seq_size_bytes);
        tmp_host_elem->text = seq1;
        tmp_host_elem->pattern = seq2;
        tmp_host_elem->len = this->sequence_len;
    }
    hipMemcpy(this->d_elements, tmp_wf_elements_host,
               this->num_elements * sizeof(WF_element), hipMemcpyHostToDevice);
    CUDA_CHECK_ERR;
    free(tmp_wf_elements_host);

#ifdef DEBUG_MODE
    size_t total_memory  = req_memory + seq_size_bytes * 2 * this->num_elements;
    DEBUG("GPU pattern/text memory initialized, %zu MiB used.",
          total_memory / (1 << 20));
    CLOCK_STOP("GPU pattern/text memory initializaion.")
#endif

    // Start the clock for benchmark in DEBUG_MODE
    CLOCK_START()

    // Create offsets into the GPU
    req_memory = this->num_elements * sizeof(edit_wavefronts_t);
    if (req_memory > MAX_GPU_SIZE) {
        WF_ERROR("Required memory is bigger than available memory in GPU");
        return false;
    }
    hipMalloc((void **) &(this->d_wavefronts), req_memory);
    CUDA_CHECK_ERR;
    hipMemset((void *) this->d_wavefronts, 0, req_memory);
    CUDA_CHECK_ERR;

    size_t offset_size = 2 * this->max_distance * sizeof(ewf_offset_t);
    for (int i=0; i<this->num_elements; i++) {
        // A temporary CPU edit_wavefront_t is needed. As we can not access
        // pointers inside GPU, the hipMalloc result is saved on host RAM, and
        // then sent to device.
        edit_wavefront_t tmp_host_wf = {0};
        hipMalloc((void **) &(tmp_host_wf.offsets), offset_size);
        CUDA_CHECK_ERR;
        hipMemcpy(&this->d_wavefronts[i].wavefront, &tmp_host_wf,
                   sizeof(edit_wavefront_t), hipMemcpyHostToDevice);
        CUDA_CHECK_ERR;
        hipMalloc((void **) &(tmp_host_wf.offsets), offset_size);
        CUDA_CHECK_ERR;
        hipMemcpy(&this->d_wavefronts[i].next_wavefront, &tmp_host_wf,
                   sizeof(edit_wavefront_t), hipMemcpyHostToDevice);
        CUDA_CHECK_ERR;
    }

#ifdef DEBUG_MODE
    total_memory = req_memory + offset_size * this->num_elements * 2;
    DEBUG("GPU offsets memory initialized, %zu MiB used.", total_memory / (1 << 20));
    CLOCK_STOP("GPU offsets memory initialization.")
#endif

    return true;
}

bool Sequences::GPU_memory_free () {
    DEBUG("Freeing GPU memory.");
    CLOCK_INIT()
    CLOCK_START()
    // Free all the texts/patterns
    WF_element tmp_host_elem = {0};
    hipMemcpy(&tmp_host_elem, this->d_elements,
               sizeof(WF_element), hipMemcpyDeviceToHost);
    CUDA_CHECK_ERR;
    hipFree(tmp_host_elem.text);
    CUDA_CHECK_ERR;
    hipFree(this->d_elements);
    CUDA_CHECK_ERR;

    CLOCK_STOP("Text/patterns GPU memory freed.")

    // Free all the offsets
    for (int i=0; i<this->num_elements; i++) {
        edit_wavefront_t tmp_host_wf = {0};
        hipMemcpy(&tmp_host_wf, &this->d_wavefronts[i].wavefront,
                   sizeof(edit_wavefront_t), hipMemcpyDeviceToHost);
        CUDA_CHECK_ERR;
        hipFree(tmp_host_wf.offsets);
        CUDA_CHECK_ERR;
        hipMemcpy(&tmp_host_wf, &this->d_wavefronts[i].next_wavefront,
                   sizeof(edit_wavefront_t), hipMemcpyDeviceToHost);
        CUDA_CHECK_ERR;
        hipFree(tmp_host_wf.offsets);
        CUDA_CHECK_ERR;
    }
    hipFree(d_wavefronts);
    CUDA_CHECK_ERR;

    DEBUG("GPU memory freed.")
    return true;
}
